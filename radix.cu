#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <time.h>
#include <stdlib.h>

__global__ void histogramKernel(int * inArray, int * outArray, int * radixArray, int arrayLength, int significantDigit){

	int index 	= blockIdx.x * blockDim.x + threadIdx.x;

	int radix;
	int arrayElement;
	
	if(index < (arrayLength))
	{	
		arrayElement 			= inArray[index];
		radix				= ((arrayElement/significantDigit) % 10);
		radixArray[index]		= radix;
		
		printf("\tArray Element : %d\tRadix in Histogram Array : %d\n", arrayElement, radix);	
		atomicAdd(&outArray[radix], 1);
	}
}

__global__ void indexArrayKernel(int * radixArray,  int * bucketArray, int * indexArray, int arrayLength, int significantDigit){
	
	int index 	= blockIdx.x * blockDim.x + threadIdx.x;

	int i;
	int radix;
	int pocket;
	
	if(index < 10){
		for(i = 0; i < arrayLength; i++){
	
			radix			= radixArray[arrayLength -i -1];
			if(radix == index){
				pocket				= --bucketArray[radix];
				printf("\tIndex : %d\tBucket Array[%d] : %d\tRadix : %d\t Pocket : %d\n", index, radix, bucketArray[radix], radix, pocket);			
				indexArray[arrayLength -i -1] 	= pocket;		
			}
		}
	}
}

__global__ void semiSortKernel(int * inArray, int * outArray, int* indexArray, int arrayLength, int significantDigit){

	int index 	= blockIdx.x * blockDim.x + threadIdx.x;

	int arrayElement;
	int arrayIndex;

	if(index < arrayLength){
		arrayElement			= inArray[index];
		arrayIndex 			= indexArray[index];
	
		outArray[arrayIndex]		= arrayElement;
	}
	
	

}

void printArray(int * array, int size){
	int i;
	printf("[ ");
	for (i = 0; i < size; i++)
		printf("%d ", array[i]);
	printf("]\n");
}

int findLargestNum(int * array, int size){
	int i;
	int largestNum = -1;
	for(i = 0; i < size; i++){
		if(array[i] > largestNum)
			largestNum = array[i];
	}
	return largestNum;
}

void cudaScanThrust(int* inarray, int arr_length, int* resultarray) {

    	int length = arr_length;
    
	thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    	thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    	hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    	thrust::inclusive_scan(d_input, d_input + length, d_output);

    	hipDeviceSynchronize();

    	hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    	thrust::device_free(d_input);
    	thrust::device_free(d_output);
}

void radixSort(int * array, int size){
	printf("\n\nRunning Radix Sort on Unsorted List!\n\n");

	int significantDigit 	= 1;
	int largestNum 		= findLargestNum(array, size);

	int * inputArray;
	int * outputArray;
	int * radixArray;
	int * bucketArray;
	int * indexArray;
	int * semiSortArray;


	hipMalloc((void **)& inputArray, sizeof(int)*size);
	hipMalloc((void **)& indexArray, sizeof(int)*size);
	hipMalloc((void **)& radixArray, sizeof(int)*size);
	hipMalloc((void **)& outputArray, sizeof(int)*size);
	hipMalloc((void **)& semiSortArray, sizeof(int)*size);
	hipMalloc((void **)& bucketArray, sizeof(int)*10);
	

	int radixSortArray[100000];	
	while (largestNum / significantDigit > 0){
		printf("\tSorting: %d's place ", significantDigit);
		printArray(array, size);
		
		int threadCount;
		int blockCount;
	
		threadCount 			= 256;
		blockCount 			= (size-1)/threadCount +1;

		
		int bucket[10] = { 0 };
		hipMemcpy(inputArray, array, sizeof(int)*size, hipMemcpyHostToDevice);
		hipMemcpy(bucketArray, bucket, sizeof(int)*10, hipMemcpyHostToDevice);
	 	
		histogramKernel<<<blockCount, threadCount>>>(inputArray, bucketArray, radixArray, size, significantDigit); 	
		hipDeviceSynchronize();
		
		
		hipMemcpy(bucket, bucketArray, sizeof(int)*10, hipMemcpyDeviceToHost);
		printf("\tBucket Array");
		printArray(bucket, 10);
		

		cudaScanThrust(bucketArray, 10, bucketArray);	
		hipMemcpy(bucket, bucketArray, sizeof(int)*10, hipMemcpyDeviceToHost);
		printf("\tBucket Array");
		printArray(bucket, 10);
		
		indexArrayKernel<<<blockCount, threadCount>>>(radixArray, bucketArray, indexArray, size, significantDigit);
		hipDeviceSynchronize();
		hipMemcpy(radixSortArray, radixArray, sizeof(int)*size, hipMemcpyDeviceToHost);
		
		printf("\tRadix Array");
		printArray(radixSortArray, size);

		semiSortKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, indexArray, size, significantDigit);
		hipDeviceSynchronize();
		hipMemcpy(array, semiSortArray, sizeof(int)*size, hipMemcpyDeviceToHost);
		
		printf("\tSorted Array");
		printArray(array, size);

		significantDigit *= 10;

		printf("\n\tBucket: ");
		printArray(bucket, 10);


	}
	
	hipFree(inputArray);
	hipFree(indexArray);
	hipFree(radixArray);
	hipFree(bucketArray);
	hipFree(outputArray);
	hipFree(semiSortArray);
}

int main(){

	printf("\n\nRunning Radix Sort Example in C!\n");
	printf("----------------------------------\n");

	int size = 100000;
	int* array;
	int i;
	int list[size];

	srand(time(NULL));

	for(i =0; i < size; i++){
		list[i]		= size -i;
	}
	
	array = &list[0];
	printf("\nUnsorted List: ");
	printArray(array, size);

	radixSort(array, size);

	printf("\nSorted List:");
	printArray(array, size);

	printf("\n");

	return 0;
}
