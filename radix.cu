#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <time.h>
#include <stdlib.h>

#include "CycleTimer.h"

#define SIZE 100000
#define THREADSIZE 256
#define BLOCKSIZE ((SIZE-1)/THREADSIZE + 1) 
#define RADIX 10

__global__ void copyKernel(int * inArray, int * semiSortArray, int arrayLength){
		
	int index 	= blockIdx.x * blockDim.x + threadIdx.x;
	
	if(index < arrayLength){
		inArray[index] 		= semiSortArray[index];
	}
}

__global__ void radixKernel(int * inArray, int* radixArray, int arrayLength, int significantDigit){

	__shared__ int inArrayShared[THREADSIZE];
	__shared__ int radixArrayShared[THREADSIZE];
	
	int index 	= blockIdx.x * blockDim.x + threadIdx.x;
	int thread 	= threadIdx.x;

	int arrayElement;
	int radix;

	if(index < arrayLength){
		inArrayShared[thread] 		= inArray[index];
	}
	
	if(index < arrayLength)
	{	
		arrayElement 			= inArrayShared[thread];
		radix				= ((arrayElement/significantDigit) % 10);
		radixArrayShared[thread]	= radix;
	}
	
	if(index < arrayLength){
		radixArray[index]		= radixArrayShared[thread];
	}

}
__global__ void histogramKernel(int * outArray, int * radixArray, int arrayLength, int significantDigit){

	__shared__ int outArrayShared[RADIX];
	
	int index 	= blockIdx.x * blockDim.x + threadIdx.x;
	int thread 	= threadIdx.x;
	int blockIndex	= blockIdx.x * RADIX;
	
	int radix;
	int i;

	if(thread ==  0){
		for(i =0; i < RADIX; i ++){
			outArrayShared[i] 		= 0;
		}
	}
	
	__syncthreads(); 

	if(index < arrayLength)
	{	
		radix				= radixArray[index];
		atomicAdd(&outArrayShared[radix], 1);
	}

	__syncthreads();
		

	if(thread == 0){
		for(i =0; i < RADIX; i++){
		
			outArray[blockIndex + i] 		= outArrayShared[i];
		}
	}
}

__global__ void combineBucket(int * blockBucketArray, int * bucketArray){
	
	__shared__ int bucketArrayShared[RADIX];

	int index 	= blockIdx.x * blockDim.x + threadIdx.x;
	
	int i;
	
	bucketArrayShared[index] 	= 0;
	
	for(i = index; i < RADIX*BLOCKSIZE; i=i+RADIX){
		atomicAdd(&bucketArrayShared[index], blockBucketArray[i]);		
	} 
	
	bucketArray[index] 		= bucketArrayShared[index];
}


__global__ void indexArrayKernel(int * radixArray,  int * bucketArray, int * indexArray, int arrayLength, int significantDigit){
	
	int index 	= blockIdx.x * blockDim.x + threadIdx.x;

	int i;
	int radix;
	int pocket;
	
	if(index < RADIX){
		
		for(i = 0; i < arrayLength; i++){
	
			radix			= radixArray[arrayLength -i -1];
			if(radix == index){
				pocket				= --bucketArray[radix];
				indexArray[arrayLength -i -1] 	= pocket;		
			}
		}
	}
}

__global__ void semiSortKernel(int * inArray, int * outArray, int* indexArray, int arrayLength, int significantDigit){

	int index 	= blockIdx.x * blockDim.x + threadIdx.x;
	
	int arrayElement;
	int arrayIndex;

	if(index < arrayLength){
		arrayElement			= inArray[index];
		arrayIndex 			= indexArray[index];
		outArray[arrayIndex]		= arrayElement;
	}
	
	

}

void printArray(int * array, int size){
	int i;
	printf("[ ");
	for (i = 0; i < size; i++)
		printf("%d ", array[i]);
	printf("]\n");
}

int findLargestNum(int * array, int size){
	int i;
	int largestNum = -1;
	for(i = 0; i < size; i++){
		if(array[i] > largestNum)
			largestNum = array[i];
	}
	return largestNum;
}


void cudaScanThrust(int* inarray, int arr_length, int* resultarray) {

    	int length = arr_length;
    
	thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    	thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    	hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    	thrust::inclusive_scan(d_input, d_input + length, d_output);

    	hipDeviceSynchronize();

    	hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    	thrust::device_free(d_input);
    	thrust::device_free(d_output);
}

void radixSort(int * array, int size){

	double startTime;
	double endTime;
	double duration;
	
	int significantDigit 	= 1;

	int threadCount;
	int blockCount;
	
	threadCount 			= THREADSIZE;
	blockCount 			= BLOCKSIZE;;
	
	int * outputArray;
	int * inputArray;
	int * radixArray;
	int * bucketArray;
	int * indexArray;
	int * semiSortArray;
	int * blockBucketArray;

	hipMalloc((void **)& inputArray, sizeof(int)*size);
	hipMalloc((void **)& indexArray, sizeof(int)*size);
	hipMalloc((void **)& radixArray, sizeof(int)*size);
	hipMalloc((void **)& outputArray, sizeof(int)*size);
	hipMalloc((void **)& semiSortArray, sizeof(int)*size);
	hipMalloc((void **)& bucketArray, sizeof(int)*RADIX);
	hipMalloc((void **)& blockBucketArray, sizeof(int)*RADIX*BLOCKSIZE);	
	
	
	hipMemcpy(inputArray, array, sizeof(int)*size, hipMemcpyHostToDevice);
	
	int largestNum;
	thrust::device_ptr<int>d_in 	= thrust::device_pointer_cast(inputArray);
	thrust::device_ptr<int>d_out;
	d_out = thrust::max_element(d_in, d_in + size);
	largestNum	 	= *d_out;	
	printf("\tLargestNumThrust : %d\n", largestNum);
	
	startTime 	= CycleTimer::currentSeconds();	
	
	int displayArray[128];

	while (largestNum / significantDigit > 0){
	
		int bucket[RADIX] = { 0 };
		hipMemcpy(bucketArray, bucket, sizeof(int)*RADIX, hipMemcpyHostToDevice);
	 	
		radixKernel<<< blockCount, threadCount>>>(inputArray, radixArray, size, significantDigit);
		hipDeviceSynchronize();
			
		histogramKernel<<<blockCount, threadCount>>>(blockBucketArray, radixArray, size, significantDigit); 	
		hipDeviceSynchronize();
		hipMemcpy(displayArray, blockBucketArray, sizeof(int)*20, hipMemcpyDeviceToHost);
		printf("\nDisplayArray: ");
		printArray(displayArray, 20);
		
	
		combineBucket<<<1, RADIX>>>(blockBucketArray,bucketArray);
		hipDeviceSynchronize(); 			
		
		cudaScanThrust(bucketArray, RADIX, bucketArray);	
		hipDeviceSynchronize();
		
		indexArrayKernel<<<blockCount, threadCount>>>(radixArray, bucketArray, indexArray, size, significantDigit);
		hipDeviceSynchronize();

		semiSortKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, indexArray, size, significantDigit);
		hipDeviceSynchronize();
			
		copyKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, size);
		hipDeviceSynchronize();
		
		
		significantDigit *= RADIX;

	}
	
	endTime		= CycleTimer::currentSeconds();
	duration	= endTime - startTime;

	hipMemcpy(array, semiSortArray, sizeof(int)*size, hipMemcpyDeviceToHost);
	
	printf("Duration : %.3f ms\n", 1000.f * duration);
	
	hipFree(inputArray);
	hipFree(indexArray);
	hipFree(radixArray);
	hipFree(bucketArray);
	hipFree(blockBucketArray);
	hipFree(outputArray);
	hipFree(semiSortArray);
}

int main(){

	printf("\n\nRunning Radix Sort Example in C!\n");
	printf("----------------------------------\n");

	int size = SIZE;
	int* array;
	int i;
	int list[size];

	srand(time(NULL));

	for(i =0; i < size; i++){
		list[i]		= SIZE -i;
	}
	
	array = &list[0];
	printf("\nUnsorted List: ");
	printArray(array, size);

	radixSort(array, size);

	printf("\nSorted List:");
	printArray(array, size);

	printf("\n");

	return 0;
}
